#include <stdio.h>
#include <deviceFunctions.h>

void printDeviceInfo()
{
	int deviceCount;

	hipGetDeviceCount(&deviceCount);

	printf("Device Count: %i\n", deviceCount);
}