#include <stdio.h>
#include <deviceFunctions.h>

void printDeviceInfo()
{
	int deviceCount;
	hipDeviceProp_t prop;
	int deviceNum = 1;

	hipGetDeviceCount(&deviceCount);
	hipGetDeviceProperties(&prop, deviceNum);

	printf("Device Count:            %i\n", deviceCount);
	printf("Global Memory Size:      %zu\n", prop.totalGlobalMem);
	printf("Shared Memory Per Block: %zu\n", prop.sharedMemPerBlock);
	printf("Max Block Dimensions:    %i\n", prop.maxThreadsPerBlock);
	printf("Max Grid Dimensions:     %i\n", prop.maxGridSize[0]);
	printf("Max Grid Dimensions:     %i\n", prop.maxGridSize[1]);
	printf("Max Grid Dimensions:     %i\n", prop.maxGridSize[2]);
	printf("Warp Size:               %i\n", prop.warpSize);
}