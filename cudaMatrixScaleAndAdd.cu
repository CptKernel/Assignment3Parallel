#include <stdio.h>
#include <hip/hip_runtime.h>
#include <matrixScaleAndAddFunctions.h>
#include <cudaMatrixScaleAndAddFunctions.h>

#define MAX_MATRIX_SIZE 50

int main(int argc, char *argv[])
{
	float a = atof(argv[1]);
	char *b = argv[2];
	char *c = argv[3];
	char *d = argv[4];

	// Error handling for input. 
	if(argc != 5)
	{
		printError(1, "4");
		return -1;
	}

	printf("Arguments: %f %s %s %s\n", a, b, c, d);
	printf("Test\n");

	int rowsB;
	int rowsC;
	int colsB;
	int colsC;
	float B[MAX_MATRIX_SIZE];
	float C[MAX_MATRIX_SIZE];
	float D[MAX_MATRIX_SIZE];

	getMatrix(&rowsB, &colsB, B, b);
	getMatrix(&rowsC, &colsC, C, c);

	int arraySizeB = rowsB * colsB;
	int arraySizeC = rowsC * colsC;

	if (arraySizeC != arraySizeB)
	{
		printError(2, "");
		return -1;
	}

	scaleAndAddKernel<<<1,1>>>(arraySizeB, a, B, C, D);
	hipDeviceSynchronize();

	for (int i = 0; i < arraySizeB; i++)
	{
		printf("%f ", D[i]);
	}
	printf("\n");

}

