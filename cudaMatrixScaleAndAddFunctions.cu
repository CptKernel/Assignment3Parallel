#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cudaMatrixScaleAndAddFunctions.h>

__global__ void scaleAndAddKernel(int size, float scale, float *B, float *C, float *D)
{
	for (int i = 0; i < size; i++)
	{
		B[i] = scale * B[i];
		printf("%f ", B[i]);
	}
	printf("\n");
}